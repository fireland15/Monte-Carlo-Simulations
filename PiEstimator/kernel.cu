#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <vector>
#include <map>
#include <iomanip>

#define ITERATIONS 125000

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void init_cudaRandStates(unsigned long seed, hiprandState* state)
{
	unsigned long long i = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(i, i % 1024 + 1 % 37, 0, &state[i]);
}

__global__ void sample_cudaRand(unsigned long long* d_sampledValues, hiprandState* state) {
	unsigned long i = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState localState = state[i];

	for (int j = 0; j < ITERATIONS; j++) {
		float x = hiprand_uniform(&localState);
		float y = hiprand_uniform(&localState);
		if ((x * x + y * y) <= 1.0f) {
			d_sampledValues[i]++;
		}
	}
	state[i] = localState;
}

int main(int argc, char** argv)
{
	int blockSize = 512;
	int gridSize = 8192 * 2;
	size_t N = blockSize * gridSize;
	unsigned long long *v = new unsigned long long[N];

	unsigned long long *d_out;
	gpuErrchk(hipMalloc((void**)&d_out, N * sizeof(unsigned long long)));
	std::cout << "Allocated " << N * sizeof(unsigned long long) << " bytes for output values" << std::endl;
	gpuErrchk(hipMemset(d_out, 0, N));

	hiprandState *d_state;
	gpuErrchk(hipMalloc((void**)&d_state, N * sizeof(hiprandState)));
	std::cout << "Allocated " << N * sizeof(hiprandState) << " bytes for hiprand state" << std::endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// generate random numbers
	init_cudaRandStates<<<gridSize, blockSize>>>(6, d_state);
	hipEventRecord(stop);
	gpuErrchk(hipDeviceSynchronize());

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Random Number Generator states initialized in " << milliseconds << "ms" << std::endl;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	sample_cudaRand<<<gridSize, blockSize>>>(d_out, d_state);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "PI Estimation Complete in " << milliseconds << "ms" << std::endl;

	gpuErrchk(hipMemcpy(v, d_out, N * sizeof(unsigned long long), hipMemcpyDeviceToHost));
	
	/*std::vector<float> uniformSample(v, v + (N));
	std::map<int, int> histogram;

	for (unsigned int i = 0; i < uniformSample.size(); i++) {
		float value = uniformSample[i];
		int bin = floor(value * 100.0f);
		if (histogram.count(bin) == 0) {
			histogram[bin] = 1;
		}
		else {
			histogram[bin]++;
		}
	}*/

	std::vector<unsigned long long> results(v, v + N);
	unsigned long long totalInRadius = 0;
	for (unsigned int i = 0; i < results.size(); i++) {
		totalInRadius += results[i];
	}
	unsigned long long totalPoints = N * ITERATIONS;

	std::cout << "in: " << totalInRadius << std::endl;
	std::cout << "all: " << totalPoints << std::endl;
	std::cout << "pi: " << std::setprecision(10) << 4.0 * totalInRadius / long double(totalPoints) << std::endl;

	/*std::ofstream f("results.csv", std::ios::out);
	if (f.is_open()) {
		for (auto kvp : histogram) {
			f << kvp.first / 100.0f << "," << kvp.second << ",\n";
		}

		f.close();
	}
	else {
		std::cout << "Failed to open 'results.csv'" << std::endl;
	}*/

	hipFree(d_out);
	delete[] v;

	return 0;
}