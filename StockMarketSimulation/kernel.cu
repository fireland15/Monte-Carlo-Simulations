#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <fstream>

#include "DataCollection.hpp"

int main(int argc, char** argv) {
	if (argc <= 1) {
		std::cout << "Input files required in arguments" << std::endl;
		return 0;
	}

	std::string filename = argv[1];

	std::fstream f(filename);
	DataCollection data = DataCollection::LoadFromCsvFormattedStream(f);
}